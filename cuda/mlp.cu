#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 

typedef struct 
  {
  void *device_pointer;
  int rows;
  int columns;
  } memory_cell;

memory_cell *memory_cells = (memory_cell *)0;

extern "C" {
void init_memory_cells(int how_many)
  {
  int i;
  memory_cells = (memory_cell *)malloc(sizeof(memory_cell) * how_many);
  for(i = 0; i < how_many; i++) 
    {
    memory_cells[i].device_pointer = (void *)0;
    }
  }
}

typedef struct
  {
  int input_units;
  int hidden_units;
  int output_units;
  } network;

/* first figure out the capacity of the device, we want to use as much memory as possible, without necessarily touching the bus at all */

void handleCudaErrors(hipError_t err)
  {
  if (err != hipSuccess)
    {
    printf("CUDA ERROR: %s\n", hipGetErrorString(err));
    }
  }

void get_device_statistics( hipDeviceProp_t *deviceProp )
  {
  hipSetDevice(0);
  hipGetDeviceProperties(deviceProp, 0);
  };

/* then we will need a block of memory for data, and one for weights */
typedef struct
  {
  void *layer1;
  void *layer2;
  void *data;
  } mem_map;

extern "C" {
void alloc_one_mem(int cell, int height, int width)
  {
  handleCudaErrors ( hipMalloc
    (
    (void **) &(memory_cells[cell].device_pointer), 
    height * width * sizeof(float) 
    ));
  memory_cells[cell].columns = width;
  memory_cells[cell].rows = height;
  printf("%x\n", memory_cells[cell].device_pointer);
  }
}

void alloc_all_the_mems
  ( 
  mem_map *mmap, 
  network network,
  int num_records, 
  hipDeviceProp_t deviceProp 
  )
  {
  /*memory is allocated on 256 byte boundaries
    TODO: how consistent is this*/
  /*input * hidden for the first layer of weights*/
  
  handleCudaErrors ( hipMalloc
    (
    (void **) &(mmap->layer1), 
    network.input_units * network.hidden_units * sizeof(float) 
    ));
  /*hidden * output for the second layer of weights*/
  handleCudaErrors ( hipMalloc
    (
    (void **) &(mmap->layer2), 
    network.hidden_units * network.input_units * sizeof(float) 
    ));
  handleCudaErrors ( hipMalloc
    (
    (void **) &(mmap->data), 
    num_records * (network.input_units + network.output_units) 
      * sizeof(float) 
    )); 
  }

extern "C" {
void load_mem(int cell, float *mem)
  {
  handleCudaErrors ( hipMemcpy 
    ( 
    memory_cells[cell].device_pointer, mem,
    memory_cells[cell].rows*memory_cells[cell].columns*sizeof(float), 
    hipMemcpyHostToDevice 
    ));
  }

void unload_mem(int cell, float **ptr, int *rows, int *columns)
  {
  float *ret = (float *)malloc(sizeof(float)*memory_cells[cell].columns*memory_cells[cell].rows);
  handleCudaErrors ( hipMemcpy
    (
    ret,
    memory_cells[cell].device_pointer,
    memory_cells[cell].rows*memory_cells[cell].columns*sizeof(float), 
    hipMemcpyDeviceToHost 
    ));
  *ptr = ret;
  *rows = memory_cells[cell].rows;
  *columns = memory_cells[cell].columns;
  }
}

extern "C" {
void unload_row(int cell, int row_index, float **ptr, int *columns)
  {
  float *ret = (float *)malloc(sizeof(float)*memory_cells[cell].columns);
  handleCudaErrors ( hipMemcpy
    (
    ret,
    (float *)memory_cells[cell].device_pointer + 
      (memory_cells[cell].columns * row_index),
    memory_cells[cell].columns*sizeof(float), 
    hipMemcpyDeviceToHost 
    ));
  *ptr = ret;
  *columns = memory_cells[cell].columns;
  }
}


extern "C" {
void init_weights_from_sampler(int cell_id, float (*sampler)())
  {
  int height, width, i;
  float *host_memory;
  height = memory_cells[cell_id].rows;
  width = memory_cells[cell_id].columns;
  host_memory = (float *)malloc(sizeof(float)*height*width);
  for(i = 0; i < height*width; i++) {host_memory[i] = sampler();};
  load_mem(cell_id, host_memory);
  free(host_memory); 
  }
}

/*we need to initialize the weight matrix the first time*/
void init_weights_without_file(network net, mem_map mmap)
  {
  float layer1[net.input_units*net.hidden_units];
  float layer2[net.hidden_units*net.output_units];
  int i;
  for(i = 0; i < net.input_units*net.hidden_units; i++)
    {
    layer1[i] = (float)rand()/RAND_MAX;
    }
  for(i = 0; i < net.hidden_units*net.output_units; i++)
    {
    layer2[i] = (float)rand()/RAND_MAX;
    }
  handleCudaErrors ( hipMemcpy 
    ( 
    mmap.layer1, &layer1, 
    net.input_units*net.hidden_units, hipMemcpyHostToDevice 
    ));
  handleCudaErrors ( hipMemcpy 
    ( 
    mmap.layer2, &layer2, 
    net.hidden_units*net.output_units, hipMemcpyHostToDevice 
    ));
 
  }

/*we need to save the file*/
void save_weights(network net, mem_map mmap)
  {
  float layer1[net.input_units*net.hidden_units];
  float layer2[net.hidden_units*net.output_units];
  FILE *fp;
  handleCudaErrors ( hipMemcpy
    (
    &layer1,
    mmap.layer1,
    net.output_units*net.hidden_units, hipMemcpyDeviceToHost 
    ));
  handleCudaErrors ( hipMemcpy
    (
    &layer2,
    mmap.layer2,
    net.hidden_units*net.output_units, hipMemcpyDeviceToHost 
    ));
  fp = fopen("save.file", "w");
  fwrite(layer1, sizeof(float), net.input_units*net.hidden_units, fp);
  fwrite(layer2, sizeof(float), net.hidden_units*net.output_units, fp);
  fclose(fp);
  }

/*we need to initialize the weights from a file*/
void init_weights_with_file(FILE *fp, network net, mem_map mmap)
  {
  float layer1[net.input_units*net.hidden_units];
  float layer2[net.hidden_units*net.output_units];
  handleCudaErrors ( hipMemcpy 
    ( 
    mmap.layer1, &layer1, 
    net.input_units*net.hidden_units, hipMemcpyHostToDevice 
    ));
  handleCudaErrors ( hipMemcpy 
    ( 
    mmap.layer2, &layer2, 
    net.hidden_units*net.output_units, hipMemcpyHostToDevice 
    ));
  fread(layer1, sizeof(float), net.input_units*net.hidden_units, fp);
  fread(layer2, sizeof(float), net.hidden_units*net.output_units, fp);
  fclose(fp);
  }

/*we need to load a block of data (can we do this while the kernel is working?*/
void load_data(void *fp, mem_map *mmap)
  {
  }

__global__ void matrix_multiply_kernel(int dim1, float *matA, int dim2, float *matB, int dim3, float *matOut)
  {
  /*matOut[x,y] = sum over dim2 matA[x, i] * matB[i, y]*/
  /* the dataset will be row major so the other matrices should be too

     (0,0) (0,1) (0,2) * (0,0) =  
                         (1,0) 
                         (2,0) */
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  float acc = 0.0;
  int i;
  if(myX < dim3 && myY < dim1) for(i = 0; i < dim2; i++)
    { 
    acc += matA[myY*dim2+i]*matB[i*dim3+myX];
    }
    matOut[myY*dim3+myX] = acc;
  }

__global__ void rectify_kernel(float *array, int rows, int columns)
  {
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  if(myX < columns && myY < rows && array[myY*columns + myX] <= 0.0)
    array[myY*rows + myX] = 0.0;
  }

extern "C" {
void rectify(int cell)
  {
  int h, w;
  float *p;
  h = memory_cells[cell].rows;
  w = memory_cells[cell].columns;
  p = (float *)memory_cells[cell].device_pointer;
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(h/block_width);
  grid_height=ceil(w/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  rectify_kernel<<<dimBlock, dimGrid>>>(p, h, w);  
  }
}

__global__ void bp_rectify(float *weights, float *acts, float *prior_losses, int inputs, int outputs, int samples, float *losses)
  {
  /*
  weights is the weights matrix, its dimension is (inputs, outputs)
  acts is the activations at the output layer, its dimension is (samples, outputs)
  prior_losses is the losses measured at the output layer, it's dimension is (samples, outputs)
  loss is the loss measured at the input layer, its dimension is (samples, inputs)
  */
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  float weight, act, loss, derv;
  int i;
  if(myX < inputs && myY < samples )
    {
    weight = weights[myY*outputs + myX];
    act = acts[myY*inputs + myX];
    derv = 0.0?(act <= 0.0):1.0;
    loss = 0;
    for(i = 0; i < outputs; i++)
      {
      /*the weights matrix will be for outgoing weights,
        so the number of rows will be the size of the output
        the number of columns will match the total number
        of activations. So we are summing over columns, myY indicates
        the sample we are looking at 
      */
      loss = loss + weight*derv*prior_losses[myY*outputs + i];
      }
    losses[myY*inputs + myX] = loss;
    }
  }

__global__ void bp_softmax_cross_entropy_kernel(float *probs, float *truths, int columns, int rows, float *loss)
  {
  /*want to compute the derivative of the error E 
    with respect to the weights W
    E = crossentropy(softmax(sum(W*inputs)), truth)
    d/dx crossentripy(softmax(x), truth) = x-truth ;; (what?)
    so dE/dW = (x-truth) * d/dW (sum(W*inputs)) = 
    (x-truth) * input
  */
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  float prob, truth;
  if(myX < columns && myY < rows)
    {
    prob = probs[myY*columns + myX];
    truth = truths[myY*columns + myX];
    loss[myY*columns + myX] = prob-truth;
    }
  }

__global__ void sum_over_activations_kernel(float *activations, float *coefficients, int columns, int rows, int sample_size, float *output)
  {
  /*coefficients contains a series of errors at the output layer from
    backprop (sample_size x outputs) ;
    activations contains the activations of the 
    previous layer (sample_size x inputs) ;
    for the result I need to multiply each coefficient by its associated
    activation. All coefficients times all activations for a 
    (inputs x outputs) matrix with which to adjust the weight matrix
    these will be summed for the output 
    the rows are */
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  w = memory_cells[activations].columns;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  int i;
  float acc;
  acc = 0.0;
  if(myX < columns && myY < rows)
    {
    for(i = 0; i < sample_size; i++)
      {
      acc = acc + 
        (activations[rows * i + myY] *
         coefficients[columns * i + myX])
      }
    output[myY*columns + myX] = acc/sample_size;
    }
  } 

extern "C" {
void bp_softmax_cross_entropy(int probs, int truths, int loss)
  {
  int h, w;
  float *p, t, l;
  h = memory_cells[probs].rows;
  w = memory_cells[probs].columns;
  if(memory_cells[truths].columns != w){printf("expected %d columns of truth\n", w); return;}
  if(memory_cells[truths].rows != h){printf("expected %d rows of truth\n", h); return;}
  if(memory_cells[loss].rows != h) {printf("expected %d rows for loss\n", h); return;}
  if(memory_cells[loss].columns != w) {printf("expected %d rows for loss\n", w; return;}
  p = (float *)memory_cells[probs].device_pointer;
  t = (float *)memory_cells[truths].device_pointer;
  l = (float *)memory_cells[loss].device_pointer;
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(h/block_width);
  grid_height=ceil(w/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  bp_softmax_cross_entropy_kernel<<<dimBlock, dimGrid>>>(p, t, h, w, l);  
  }
}

extern "C" {
void sum_over_activations()
  {
  int h, w;
  float *p, t, l;
  h = memory_cells[probs].rows;
  w = memory_cells[probs].columns;
  if(memory_cells[truths].columns != w){printf("expected %d columns of truth\n", w); return;}
  if(memory_cells[truths].rows != h){printf("expected %d rows of truth\n", h); return;}
  if(memory_cells[loss].rows != h) {printf("expected %d rows for loss\n", h); return;}
  if(memory_cells[loss].columns != w) {printf("expected %d rows for loss\n", w; return;}
  p = (float *)memory_cells[probs].device_pointer;
  t = (float *)memory_cells[truths].device_pointer;
  l = (float *)memory_cells[loss].device_pointer;
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(h/block_width);
  grid_height=ceil(w/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  bp_softmax_cross_entropy_kernel<<<dimBlock, dimGrid>>>(p, t, h, w, l);  
    
  }
}

__global__ void update_weights(float *activations, float *weights, float *losses, int rows, int columns, float epsilon)
  {
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  int myIdx = myY*columns + myX;
  if(myX < columns && myY < rows)
    {
    weights[myIdx] = weights[myIdx] + (
                     activations[myIdx] *
                     losses[myIdx] *
                     epsilon );
    }
  }

__global__ void softmax_kernel(float *activations, int rows, int columns, float *partition_function)
  {
  int myX = blockIdx.x*blockDim.x + threadIdx.x;
  int myY = blockIdx.y*blockDim.y + threadIdx.y;
  float act;
  if(myX < columns && myY < rows)
    {
    act = activations[myY*columns + myX];
    act = act/partition_function[myY];
    activations[myY*columns + myX] = act;
    }
  }

extern "C" {
void softmax(int activations, int partition_function)
  {
  int h, w;
  float *acts, *part;
  h = memory_cells[activations].rows;
  w = memory_cells[activations].columns;
  if(memory_cells[partition_function].columns != 1){printf("expected only 1 column of output\n"); return;}
  if(memory_cells[partition_function].rows != h) {printf("expected %d rows for output got %d\n", h, memory_cells[partition_function].rows); return;}
  acts = (float *)memory_cells[activations].device_pointer;
  part = (float *)memory_cells[partition_function].device_pointer;
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(h/block_width);
  grid_height=ceil(w/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  softmax_kernel<<<dimBlock, dimGrid>>>(acts, h, w, part);  
   
  }
}

__global__ void partition_function_kernel(float *activations, int rows, int columns, float *partition_function)
  {
  int myIdx = blockIdx.x*blockDim.x+threadIdx.x;
  int i;
  float part = 0.0;
  for(i = 0; i < columns; i++)
    {
    part = part + exp(activations[myIdx*columns + i]);
    }
  partition_function[myIdx] = part;
  }


extern "C" {
void partition_function(int inp, int outp)
  {
  int h, w;
  float *pin, *pout;
  h = memory_cells[inp].rows;
  w = memory_cells[outp].columns;
  if(memory_cells[outp].columns != 1){printf("expected only 1 column of output\n"); return;}
  if(memory_cells[outp].rows != h) {printf("expected %d rows for output got %d\n", h, memory_cells[outp].rows); return;} 
  pin = (float *)memory_cells[inp].device_pointer;
  pout = (float *)memory_cells[outp].device_pointer;
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(h/block_width);
  grid_height=ceil(w/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  partition_function_kernel<<<dimBlock, dimGrid>>>(pin, h, w, pout);  
  }
}

__global__ void cross_entropy_loss_kernel
  (
  float *model,
  float *actual,
  int rows,
  int columns,
  float *loss
  )
  {
  int myX = blockIdx.x*blockDim.x+threadIdx.x;
  int myY = blockIdx.y*blockDim.y+threadIdx.y;
  float a, m, l;
  m = model[myY*columns + myX];
  a = actual[myY*columns + myY];
  l = (a)*log(m) + (1-a)*log(1-m);
  loss[myY*columns + myX] = l;
  }

extern "C" {
void cross_entropy_loss(int model, int actual, int loss)
  {
  int h, w;
  float *mod, *act, *l;
  h = memory_cells[model].rows;
  w = memory_cells[model].columns;
  /*all inputs need the same dimensions*/
  if(memory_cells[loss].columns != w){printf("expected %d column of loss got %d\n", w, memory_cells[loss].columns); return;}
  if(memory_cells[loss].rows != h) {printf("expected %d rows for loss got %d\n", h, memory_cells[loss].rows); return;} 
  if(memory_cells[actual].rows != h) {printf("expected %d rows for actuals got %d\n", h, memory_cells[actual].rows); return;} 
  if(memory_cells[actual].columns != w) {printf("expected %d columns for actuals got %d\n", h, memory_cells[actual].columns); return;} 
  mod = (float *)memory_cells[model].device_pointer;
  act = (float *)memory_cells[actual].device_pointer;
  l = (float *)memory_cells[loss].device_pointer;
  
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(h/block_width);
  grid_height=ceil(w/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  cross_entropy_loss_kernel<<<dimBlock, dimGrid>>>(mod, act, h, w, l);  
  
  }
}

__device__ void forward_pass()
  {
  /*two matrix multiplications a relu and a softmax*/
  }

__device__ void backward_pass()
  {
  /*measure error using Kulbach-Lieber divergence
    first layer error = */
  }

__device__ void update_weights()
  {
  }

/*this will run on the device :
  a forward pass to compute error
  a backward pass to compute gradient
  an update to the weights
must they all run in the same kernel function?
*/
__global__ void do_the_things()
  {
  forward_pass();
  }
extern "C" {
void matrix_multiply(int cellA, int cellB, int cellC)
  {
  int d1 = memory_cells[cellA].rows; /*should == cellC rows*/
  int d2 = memory_cells[cellB].rows; /*should == cellA columns*/
  int d3 = memory_cells[cellB].columns; /*should == cellC columns*/

  if(d1 != memory_cells[cellC].rows) {printf("Wrong number of output rows expected %d got %d\n", d1, memory_cells[cellC].rows); return; }
  if(d2 != memory_cells[cellA].columns) {printf("Wrong number of columns in first param expected %d got %d\n", d2, memory_cells[cellA].columns); return; }
  if(d3 != memory_cells[cellC].columns) {printf("Wrong number of output columns expected %d got %d\n", d3, memory_cells[cellC].rows); return; }

  float *addrA = (float *)memory_cells[cellA].device_pointer;
  float *addrB = (float *)memory_cells[cellB].device_pointer;
  float *addrC = (float *)memory_cells[cellC].device_pointer;

  /*on NVS 3100M device block can have max 512 threads
    grid can have max 65535 blocks in any one dimension
    I need d1 x d3 threads total, d1 will typically be very large
    the requirements are
    grid_width * block_width >= d3 (as close as possible) 
    grid_height * grid_width >= d1 (as close as possible) 
    thread block dimensions should be multiples of 32.
    I am not using block shared memory so they might as well
    be small.
    */
  int block_width, block_height, grid_width, grid_height;
  block_width=32;
  block_height=32;
  grid_width=ceil(d1/block_width);
  grid_height=ceil(d3/block_height);
  /*end block size thoughts*/
  dim3 dimBlock(block_width, block_height, 1);
  dim3 dimGrid(grid_width, grid_height ,1);
  printf("%d %d\n", d1, d3);
  matrix_multiply_kernel<<<dimBlock, dimGrid>>>(d1, addrA, d2, addrB, d3, addrC);
  }
}

int old_main(int argc, char *argv[])
  {
  hipDeviceProp_t dev;
  mem_map mmap;
  network nn;
  FILE *fp;
  nn.input_units = 1;
  nn.hidden_units = 1;
  nn.output_units = 1;

  get_device_statistics(&dev);
  alloc_all_the_mems(&mmap, nn, 100, dev);
  printf("%x %x %x\n", mmap.layer1, mmap.layer2, mmap.data);
  fp = fopen("save.file", "r");
  if(fp == 0)
    {
    printf("weights file not found\n");
    init_weights_without_file(nn, mmap);
    } else {
    printf("found existing weights file\n");
    init_weights_with_file(fp, nn, mmap);
    }
  /*do_the_things();*/
  save_weights(nn, mmap);
  return 0;
  }
